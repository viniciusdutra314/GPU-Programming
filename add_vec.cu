#include <random>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add_vec(float* v,float * u,float* result){
    auto index=threadIdx.x;
    result[index]=v[index]+u[index];
}

int main(){
    int N=100;
    int size=sizeof(float)*N;
    float h_v[N],h_u[N];
    float *d_v,*d_u,*d_result;
    float *h_result=(float*) malloc(size);
    for (int i=0;i<N;i++){
        h_v[i]=0.75*i;
        h_u[i]=0.25*i;
    }
    hipMalloc(&d_v,size);
    hipMalloc(&d_u,size);
    hipMalloc(&d_result,size);
    
    hipMemcpy(d_v,h_v,size,hipMemcpyHostToDevice);
    hipMemcpy(d_u,h_u,size,hipMemcpyHostToDevice);
    add_vec<<<1,N>>>(d_v,d_u,d_result);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel Error: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();
    

    hipMemcpy(h_result,d_result,size,hipMemcpyDeviceToHost);
    
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_result);
    
    
    for (int i=0;i<N;i++){
        printf("%f ",h_result[i]);
    }
    printf("\n");


}