#include <random>
#include <iostream>
#include <hip/hip_runtime.h>
#include <exception>

__global__ void add_matrix(float* A,float * B,float* C,int N){
    int index_x=blockIdx.x*blockDim.x+threadIdx.x;
    int index_y=blockIdx.y*blockDim.y+threadIdx.y;
    int index_array=index_y*N +index_x;
    C[index_array]=A[index_array]+B[index_array];
}

int main(){
    int N=16;
    int size=sizeof(float)*N*N;
    float h_A[N][N],h_B[N][N];
    float *d_A,*d_B,*d_C;
    for (int i=0;i<N;i++){
        for (int j=0;j<N;j++){
            float x=(float) rand()/RAND_MAX;
            h_A[i][j]=1-x;
            h_B[i][j]=x;
        };
    };
    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);
    
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
    int numBlocks = 1;
    dim3 threadsPerBlock(N, N);
    add_matrix<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C,N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel Error: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();
    
    float *h_result=(float*) malloc(size);
    hipMemcpy(h_result,d_C,size,hipMemcpyDeviceToHost);
    
    hipFree(d_B);
    hipFree(d_A);
    hipFree(d_C);
    
    
    for (int i=0;i<N*N;i++){
        if (h_result[i]!=1){
            printf("%f ",h_result[i]);
        };
    }
    printf("\n");


}